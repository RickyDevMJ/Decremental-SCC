#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/set_operations.h>
#include <thrust/execution_policy.h>

// todo : implement dyn_graph data-structure to represent graphs

// data-structure to store a node of the SCC-Tree
class scc_tree {
public:

    // unique identifier for every node
    long id;
    // The graph represented by the node, NULL if it is leaf
    dyn_graph G;
    // parent pointer of the node in the tree
    scc_tree *parent;
    // vector like representation of children of current node
    long max_children, num_children;
    scc_tree **children;
    long size;
    // this array stores all the vertices of base graph contained in the current node
    long *vertices;
};

// simple structure to hold the unreachable vertices and edges; return value of find_unreachable
struct unreachable
{
	thrust::device_vector<long> U;
	thrust::device_vector<std::pair<long, long>> I;
};

// simple function to calaculate depth of a node from root of the tree
long depth(scc_tree *node)
{
	long d = -1;
	while (node)
	{
		d++;
		node = node->parent;
	}
	return d;
}

// simple function to find least common ancestor of two nodes in the tree
scc_tree *LCA(scc_tree *n1, scc_tree *n2)
{
	long d1 = depth(n1), d2 = depth(n2);
	long diff = d1 - d2;

	// If n2 is deeper, swap n1 and n2
	if (diff < 0)
	{
		scc_tree *temp = n1;
		n1 = n2;
		n2 = temp;
		diff = -diff;
	}

	// Move n1 up until it reaches the same level as n2
	while (diff--)
		n1 = n1->parent;

	while (n1 && n2)
	{
		if (n1 == n2)
			return n1;
		n1 = n1->parent;
		n2 = n2->parent;
	}

	return NULL;
}

// kernel corresponding to the helper function lift_up
__global__ void lift_up_kernel(scc_tree *T, unreachable &R) {

	long tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < T->num_children) {
		if (T->children[tid] == NULL)
				return;

		for (long j = 0; j < R.U.size(); j++) {
			if (T->children[tid] == t_array[R.U[j]]) {
				t_array[R.U[j]]->parent = T->parent;

				T->children[tid] = NULL;
				atomicSub(&T->num_children, 1);

				// todo : modify T.G

				if (T->parent != NULL)
					// todo : modify T->parent and T->parent.G

				break;
			}
		}
	}
}

// helper function to recursively lift up the deleted node in the SCC-tree
void lift_up(scc_tree *T, unreachable &R, scc_tree **t_array) {

	unreachable *d_R;
	hipMalloc((void **)&d_R, sizeof(unreachable));
	hipMemcpy(d_R, &R, sizeof(unreachable), hipMemcpyHostToDevice);

	long nthreads = BLOCK_SIZE;
	long nblocks = (T->max_children - 1) / nthreads + 1;

	// todo: transfer tree T to device (d_T)
	lift_up_kernel<<< nthreads, nblocks >>>(d_T, d_R);

	// if node is not the root
	if (T->parent != NULL) {
		// todo: recursive call
	}
}

// function to remove an edge from tree and subsequently from the base graph
void remove_edge(long src, long dst, long n, scc_tree **t_array) {
	scc_tree *T = LCA(t_array[src], t_array[dst]);

	// edge across two different SCCs
	if (T == NULL) {
		// todo
		return;
	}

	long S[2];
	S[0] = T->G.vertex_map[src];
	S[1] = T->G.vertex_map[dst];
	unreachable R1 = find_unreachable_down(T->G, 2, S, T->G.vertex_map[T->id]);
	unreachable R2 = find_unreachable_up(T->G, 2, S, T->G.vertex_map[T->id]);

	unreachable R;
	thrust::device_vector<long> U(R1.U.size() + R2.U.size());
	thrust::device_vector<std::pair<long, long>> I(R1.I.size() + R2.I.size());
	thrust::sort(thrust::device, R1.U.begin(), R1.U.end());
	thrust::sort(thrust::device, R2.U.begin(), R2.U.end());
	thrust::set_union(thrust::device, R1.U.begin(), R1.U.end(), R2.U.begin(), R2.U.end(), U.begin());
	thrust::sort(thrust::device, R1.I.begin(), R1.I.end());
	thrust::sort(thrust::device, R2.I.begin(), R2.I.end());
	thrust::set_union(thrust::device, R1.I.begin(), R1.I.end(), R2.I.begin(), R2.I.end(), I.begin());
	R.U = U;
	R.I = I;

	lift_up(T, R, t_array);
}

// function to find the set of vertices not reachable from the source w
unreachable find_unreachable_down(dyn_graph G, long ns, long *S, long w) {
	thrust::device_vector<long> Q[2];

	for (long i = 0; i < ns; i++) {
		long v = S[i];
		if (v != w) {
			// if v has no in-edges
			if (G.in_deg[v] == 0) {
				Q[0].push_back(v);
			}
		}
	}

	unreachable R;

	// pop from Qa and insert into Qb
	long a = 0, b = 1;
	while (!Q[a].empty()) {
		long v = Q[a][0];
		Q[a].erase(Q[a].begin());

		R.U.push_back(v);
		for (long i = G.out_row[v]; i < G.out_row[v+1]; i++) {
			R.I.push_back(v, G.out_col[i]));
		}

		long row_start = G.out_row[v], row_end = G.out_row[v+1];
		for (long i = G.out_row[v]; i < G.out_row[v+1]; i++) {
			long x = G.out_col[i];

			G.remove_edge(v, x);
			if (G.in_deg[x] == 0) {
				Q[b].push_back(x);
			}
		}

		// swap the queues
		if (Q[a].empty()) {
			long temp = a;
			a = b;
			b = a;
		}
	}

	return R;
}

// function to find the set of vertices which does not reach the sink w
unreachable find_unreachable_up(dyn_graph G, long ns, long *S, long w) {
	thrust::device_vector<long> Q[2];

	for (long i = 0; i < ns; i++) {
		long v = S[i];
		if (v != w) {
			// if v has no out-edges
			if (G.out_deg[v] == 0) {
				Q[0].push_back(v);
			}
		}
	}

	unreachable R;

	// pop from Qa and insert into Qb
	long a = 0, b = 1;
	while (!Q[a].empty()) {
		long v = Q[a][0];
		Q[a].erase(Q[a].begin());

		R.U.push_back(v);
		for (long i = G.in_row[v]; i < G.in_row[v+1]; i++) {
			R.I.push_back(std::make_pair(G.in_col[i], v));
		}

		for (long i = G.in_row[v]; i < G.in_row[v+1]; i++) {
			long x = G.in_col[i];

			G.remove_edge(x, v);
			if (G.out_deg[x] == 0) {
				Q[b].push_back(x);
			}
		}

		// swap the queues
		if (Q[a].empty()) {
			long temp = a;
			a = b;
			b = a;
		}
	}

	return R;
}
